#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>

#include "utils.h"
#include "cuda_error_check.cuh"
#include "initial_graph.hpp"
#include "parse_graph.hpp"

__global__ void pulling_kernel(std::vector<initial_vertex> * peeps, int offset, int * anyChange){

    //update me based on my neighbors. Toggle anyChange as needed.
    //offset will tell you who I am.
}

void puller(std::vector<initial_vertex> * peeps, int blockSize, int blockNum){
    /* Allocate here... */
    unsigned int *cuda_edges_src, *cuda_edges_dest, *cuda_edges_weight;
    unsigned int edges_length;
    unsigned int vertices_length = peeps.size();
    unsigned int *cuda_distance_prev = (unsigned int *) malloc(peeps.size() * sizeof(unsigned int));
    unsigned int *cuda_distance_cur = (unsigned int *) malloc(peeps.size() * sizeof(unsigned int));

    int i = 0;
    for(auto const& vertex: peeps) {
      for(auto const& edge: vertex.nbrs) {
        std::cout << "vertexIndex: " << i << " | vertexDistance: " << vertex.get_vertex_ref().distance << " | src: " << edge.srcIndex << " | dest: " << " | weight: " << edge.edgeValue.weight << endl;
      }
      i++;
    }

    setTime();

    /*
     * Do all the things here!
     **/



    std::cout << "Took " << getTime() << "ms.\n";
}
