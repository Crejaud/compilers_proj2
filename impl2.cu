#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>

#include "utils.h"
#include "cuda_error_check.cuh"
#include "initial_graph.hpp"
#include "parse_graph.hpp"

/* Work efficient edge process out of core with no shared memory */
__global__ void work_efficient_out_of_core(unsigned int edges_length,
                            unsigned int *src,
                            unsigned int *dest,
                            unsigned int *weight,
                            unsigned int *distance_prev,
                            unsigned int *distance_cur,
                            int *noChange,
                            int *is_distance_infinity_prev,
                            int *is_distance_infinity_cur,
                            int *should_update_edge) {
    unsigned int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int thread_num = blockDim.x * gridDim.x;

    unsigned int warp_id = thread_id / 32;
    unsigned int warp_num = thread_num % 32 == 0 ? thread_num / 32 : thread_num / 32 + 1;

    unsigned int load = edges_length % warp_num == 0 ? edges_length / warp_num : edges_length / warp_num + 1;
    unsigned int beg = load * warp_id;
    unsigned int end = min(edges_length, beg + load);
    unsigned int lane = thread_id % 32;
    beg += lane;
    for (unsigned int i = beg; i < end; i += 32) {
      unsigned int u = src[dataid];
      unsigned int v = dest[dataid];
      unsigned int w = weight[dataid];
      
      if (is_distance_infinity_prev[u] == TRUE) {
        continue;
      }
      //printf("%u isn't infinite distance\n", u);
      if (should_update_edge[dataid] == TRUE && distance_prev[u] + w < distance_prev[v]) {
        // relax
        //printf("%u %u\n", distance_cur[v], distance_prev[u] + w);
        unsigned int old_distance = atomicMin(&distance_cur[v], distance_prev[u] + w);
        atomicMin(&is_distance_infinity_cur[v], FALSE);
        //printf("%u %u %u %d\n", old_distance, distance_cur[v], distance_prev[u] + w, is_distance_infinity[v]);
        // test for a change!
        if (old_distance != distance_cur[v]) {
          //printf("there is change\n");
          atomicMin(noChange, FALSE);
        }
      }
    }
}

/* Work efficient edge process in core */
__global__ void work_efficient_in_core(unsigned int edges_length,
                            unsigned int vertices_length,
                            unsigned int *src,
                            unsigned int *dest,
                            unsigned int *weight,
                            unsigned int *distance,
                            int *noChange,
                            int *is_distance_infinity,
                            int *should_update_edge) {
    unsigned int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int thread_num = blockDim.x * gridDim.x;

    unsigned int iter = edges_length % thread_num == 0 ? edges_length / thread_num : edges_length / thread_num + 1;

    for (unsigned int j = 1; j < vertices_length; j++) {
      __syncthreads();
      for (unsigned int i = 0; i < iter; i++) {
        __syncthreads();
        unsigned int dataid = thread_id + i * thread_num;
        if (dataid >= edges_length)
          continue;
        unsigned int u = src[dataid];
        unsigned int v = dest[dataid];
        unsigned int w = weight[dataid];
        //printf("src %u | dest %u | weight %u | dataid %u\n", u, v, w, dataid);
        if (is_distance_infinity[u] == TRUE) {
          continue;
        }
        unsigned int temp_dist = distance[u] + w;
        if (distance[u] == -1) {
          continue;
        }
        if (temp_dist < distance[v]) {
          // relax
          //printf("%u %u\n", distance[v], temp_dist);
          int old_distance = atomicMin(&distance[v], temp_dist);
          atomicMin(&is_distance_infinity[v], FALSE);
          //printf("%u %u %u %d\n", old_distance, distance_cur[v], distance_prev[u] + w, is_distance_infinity[v]);
          // test for a change!
          if (old_distance != distance[v]) {
            //printf("there is change\n");
            atomicMin(noChange, FALSE);
          }
        }
      }
    }
}

void neighborHandler(std::vector<initial_vertex> * peeps, int blockSize, int blockNum, int sync, int smem, unsigned int *distance_cur){
  /* Will use these arrays instead of a vector
  * edges_src : array of all edges (indexed 0 to n) where the value is the vertex source index of the edge (since edges are directed)
  * edges_dest : same as above, except it tells the vertex destination index
  * edges_weight : same as above, except it tells the edge's weight
  * distance_prev : array of all vertices with their distance values
  * distance_cur : same as above
  */

  /* Allocate here... */
  unsigned int *edges_src, *edges_dest, *edges_weight;
  unsigned int edges_length = 0;
  unsigned int vertices_length = peeps->size();
  unsigned int *distance_prev = (unsigned int *) malloc(vertices_length * sizeof(unsigned int));
  int *noChange = (int *) malloc(sizeof(int));
  int *is_distance_infinity = (int *) malloc(vertices_length * sizeof(int));
  int *should_update_edge;

  *noChange = TRUE;

  unsigned int *cuda_edges_src, *cuda_edges_dest, *cuda_edges_weight;
  unsigned int *cuda_distance_prev, *cuda_distance_cur;
  int *cuda_noChange, *cuda_is_distance_infinity_prev,
    *cuda_is_distance_infinity_cur, *cuda_should_update_edge;

  // the distance to the first vertex is always 0
  distance_prev[0] = 0;
  distance_cur[0] = 0;
  is_distance_infinity[0] = FALSE;

  // setting an unsigned int to -1 will set it to the maximum value!
  for (int i = 1; i < vertices_length; i++) {
    distance_prev[i] = -1;
    distance_cur[i] = -1;
    is_distance_infinity[i] = TRUE;
  }

  // get edges_length
  for(std::vector<int>::size_type i = 0; i != vertices_length; i++) {
    edges_length += peeps->at(i).nbrs.size();
  }

  // malloc edges arrays
  edges_src = (unsigned int *) malloc(edges_length * sizeof(unsigned int));
  edges_dest = (unsigned int *) malloc(edges_length * sizeof(unsigned int));
  edges_weight = (unsigned int *) malloc(edges_length * sizeof(unsigned int));
  should_update_edge = (int *) malloc(edges_length * sizeof(int));


  int edge_index = 0;
  // get values for each array
  for(std::vector<int>::size_type i = 0; i != vertices_length; i++) {
    for(std::vector<int>::size_type j = 0; j != peeps->at(i).nbrs.size(); j++) {
      edges_src[edge_index] = peeps->at(i).nbrs[j].srcIndex;
      edges_dest[edge_index] = i;
      edges_weight[edge_index] = peeps->at(i).nbrs[j].edgeValue.weight;
      // initially set should_update_edges to true if the source is at 0, since everything
      // else will have infinite distance.
      if (edges_src[edge_index] == 0) {
        should_update_edge[edge_index] = TRUE;
      }
      else {
        should_update_edge[edge_index] = FALSE;
      }
      //printf("src: %u | dest: %u | weight: %u\n", edges_src[edge_index], edges_dest[edge_index], edges_weight[edge_index]);

      edge_index++;
    }
  }

  hipMalloc((void **)&cuda_edges_src, edges_length * sizeof(unsigned int));
  hipMalloc((void **)&cuda_edges_dest, edges_length * sizeof(unsigned int));
  hipMalloc((void **)&cuda_edges_weight, edges_length * sizeof(unsigned int));
  hipMalloc((void **)&cuda_distance_prev, vertices_length * sizeof(unsigned int));
  hipMalloc((void **)&cuda_distance_cur, vertices_length * sizeof(unsigned int));
  hipMalloc((void **)&cuda_noChange, sizeof(int));
  hipMalloc((void **)&cuda_is_distance_infinity_prev, vertices_length * sizeof(int));
  hipMalloc((void **)&cuda_is_distance_infinity_cur, vertices_length * sizeof(int));
  hipMalloc((void **)&cuda_should_update_edge, edges_length * sizeof(int));

  hipMemcpy(cuda_edges_src, edges_src, edges_length * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(cuda_edges_dest, edges_dest, edges_length * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(cuda_edges_weight, edges_weight, edges_length * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(cuda_distance_prev, distance_prev, vertices_length * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(cuda_distance_cur, distance_cur, vertices_length * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(cuda_noChange, noChange, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(cuda_is_distance_infinity_prev, is_distance_infinity, vertices_length * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(cuda_is_distance_infinity_cur, is_distance_infinity, vertices_length * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(cuda_should_update_edge, should_update_edge, edges_length * sizeof(unsigned int), hipMemcpyHostToDevice);

  setTime();

  /*
   * Do all the things here!
   **/
  // sync is out of core
  if (sync == 0) {
    for (unsigned int i = 1; i < vertices_length; i++) {
      //printf("pass %u\n", i);
      work_efficient_out_of_core<<<blockNum, blockSize>>>(edges_length, cuda_edges_src,
                                          cuda_edges_dest, cuda_edges_weight,
                                          cuda_distance_prev, cuda_distance_cur,
                                          cuda_noChange, cuda_is_distance_infinity_prev,
                                          cuda_is_distance_infinity_cur, cuda_should_update_edge);
      hipMemcpy(noChange, cuda_noChange, sizeof(int), hipMemcpyDeviceToHost);
      if (*noChange == TRUE) break;
      *noChange = TRUE;
      hipMemcpy(cuda_noChange, noChange, sizeof(int), hipMemcpyHostToDevice);

      // get current distance and copy it to both cuda_distance_prev and cuda_distance_cur
      hipMemcpy(distance_cur, cuda_distance_cur, vertices_length * sizeof(unsigned int), hipMemcpyDeviceToHost);
      hipMemcpy(cuda_distance_prev, distance_cur, vertices_length * sizeof(unsigned int), hipMemcpyHostToDevice);
      hipMemcpy(cuda_distance_cur, distance_cur, vertices_length * sizeof(unsigned int), hipMemcpyHostToDevice);

      hipMemcpy(is_distance_infinity, cuda_is_distance_infinity_cur, vertices_length * sizeof(unsigned int), hipMemcpyDeviceToHost);
      hipMemcpy(cuda_is_distance_infinity_prev, is_distance_infinity, vertices_length * sizeof(unsigned int), hipMemcpyHostToDevice);
      hipMemcpy(cuda_is_distance_infinity_cur, is_distance_infinity, vertices_length * sizeof(unsigned int), hipMemcpyHostToDevice);
    }
  }
  // sync is in core
  else if (sync == 1) {
    work_efficient_in_core<<<blockNum, blockSize>>>(edges_length, vertices_length,
                                        cuda_edges_src, cuda_edges_dest,
                                        cuda_edges_weight, cuda_distance_cur,
                                        cuda_noChange, cuda_is_distance_infinity_prev,
                                        cuda_should_update_edge);
  }

  else {
    // no syncing
    printf("No syncing tag\n");
    exit(1);
  }

  hipDeviceSynchronize();
  std::cout << "Took " << getTime() << "ms.\n";

  hipMemcpy(distance_cur, cuda_distance_cur, vertices_length * sizeof(unsigned int),
           hipMemcpyDeviceToHost);

  /* Deallocate. */
  hipFree(cuda_edges_src);
  hipFree(cuda_edges_dest);
  hipFree(cuda_edges_weight);
  hipFree(cuda_distance_prev);
  hipFree(cuda_distance_cur);
  hipFree(cuda_noChange);
  hipFree(cuda_is_distance_infinity_prev);
  hipFree(cuda_is_distance_infinity_cur);

  free(edges_src);
  free(edges_dest);
  free(edges_weight);
  free(distance_prev);
  free(noChange);
  free(is_distance_infinity);
}
